
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define Nblock 1024
#define Nthread 100
#define Ngrid 1
#define maxRound 5000

__global__ void setup(hiprandState *state){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(9999, index, 0, &state[index]);
}

__global__ void piEs(double *sum , int iteration, hiprandState *state)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    int count;
    for(i=iteration; i-- ;){
        double x = hiprand_uniform_double(&state[index]);
        double y = hiprand_uniform_double(&state[index]);
        if(x*x + y*y <= 1.0)
            count++;
    }
    sum[index] = 4.0 * count / iteration;
}

int main(void){
    int i;
    int iter = 100000;
    int Athread = Nblock * Nthread;
    double step = iter / Athread;
    double pi = 0.0;

    float *piSumHost = (float *)malloc(Athread * sizeof(double));
    double *piSumDev;
    hipMalloc((void**)&piSumDev, Athread);
    hipMemset(piSumDev, 0, Athread);

    hiprandState *stateDev;
    hipMalloc((void **)&stateDev, Athread);

    setup <<<Ngrid, Nblock>>> (stateDev);
    piEs  <<<Ngrid, Nblock>>> (piSumDev, iter, stateDev);

    hipMemcpy(piSumHost, piSumDev, Athread, hipMemcpyDeviceToHost);

    for(i = iter; i--;)
        pi += piSumHost[i];
    pi *= step;

    printf("Pi estimate = %.10lf\n", pi);

   free(piSumHost);
   hipFree(piSumDev);
   return 0;
}