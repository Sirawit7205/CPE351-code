//60070501054
//60070501064
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define blocksize 1024
#define gridsize 1
#define threadsize 1024

__global__ void piEstimate(long long int *countStore, int *iterations)
{
	int rank = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = 0;
	long long int count = 0;
	int itr = iterations[0];
	double x, y;
	
	hiprandState state;
	hiprand_init(rank, 0, 0, &state);
	
	while(i < itr)
	{
		x = hiprand_uniform_double(&state);	
		y = hiprand_uniform_double(&state);	

		if(((x * x) + (y * y)) <= 1.0)
			count++;
			
		i++;
	}
	countStore[rank] +=count;
}

int main(int argc, char **argv)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	int iterationsHost, *iterationsDev;
	long long int *countStoreHost, *countStoreDev;
	
	printf("Enter iterations: ");
	scanf("%d", &iterationsHost);
	
	countStoreHost = (long long int *)calloc(blocksize, sizeof(long long int));
	hipMalloc((void **)&countStoreDev, sizeof(long long int) * blocksize);
	hipMalloc((void **)&iterationsDev, sizeof(int));
	
	int i;
	
	hipMemcpy(countStoreDev, countStoreHost, sizeof(long long int) * blocksize, hipMemcpyHostToDevice);
	hipMemcpy(iterationsDev, &iterationsHost, sizeof(int), hipMemcpyHostToDevice);
	hipEventRecord(start);
	piEstimate<<<gridsize, blocksize, threadsize>>>(countStoreDev, iterationsDev);
	hipEventRecord(stop);
	hipMemcpy(countStoreHost, countStoreDev, sizeof(long long int) * blocksize, hipMemcpyDeviceToHost);
	
	float runningTime = 0;
	hipEventElapsedTime(&runningTime, start, stop);
	printf("CUDA done, took %f ms\n", runningTime);
	
	/*int max = max_per_round;
	roundHost = iterationsHost / max;
	int rem = iterationsHost % max;
	for(i = 0; i < roundHost; i++)
	{
	//printf("Start round %d\n", i);
		cudaMemcpy(countStoreDev, countStoreHost, sizeof(long long int) * blocksize, cudaMemcpyHostToDevice);
		cudaMemcpy(roundDev, &i, sizeof(int), cudaMemcpyHostToDevice);
		cudaMemcpy(iterationsDev, &max, sizeof(int), cudaMemcpyHostToDevice);
		piEstimate<<<gridsize, blocksize>>>(countStoreDev, roundDev, iterationsDev);
		cudaMemcpy(countStoreHost, countStoreDev, sizeof(long long int) * blocksize, cudaMemcpyDeviceToHost);
	}
	
	if(rem != 0)
	{
	//printf("Start rem round with %d iterations\n", rem);
		cudaMemcpy(countStoreDev, countStoreHost, sizeof(long long int) * blocksize, cudaMemcpyHostToDevice);
		cudaMemcpy(roundDev, &i, sizeof(int), cudaMemcpyHostToDevice);
		cudaMemcpy(iterationsDev, &rem, sizeof(int), cudaMemcpyHostToDevice);
		piEstimate<<<gridsize, blocksize>>>(countStoreDev, roundDev, iterationsDev);
		cudaMemcpy(countStoreHost, countStoreDev, sizeof(long long int) * blocksize, cudaMemcpyDeviceToHost);
	}*/
	
	double pi = 0.0;
	for(i = 0; i < blocksize; i++)
		pi += countStoreHost[i];

	
	long long int totalPlot = blocksize * iterationsHost;
	pi = (pi / totalPlot) * 4;
	
	printf("Pi estimate for %d iterations = %.10lf\n", iterationsHost, pi);
	
	free(countStoreHost);
	hipFree(countStoreDev);
	hipFree(iterationsDev);
	
	return 0;
}