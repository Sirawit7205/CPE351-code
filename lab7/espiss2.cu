#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define Nblock 1000
#define Nthread 1000
#define Ngrid 1

__global__ void setup(hiprandState *state){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(9999, index, 0, &state[index]);
}

__global__ void piEs(double *sum , int iteration, hiprandState *state)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    int count = 0;
    for(i=0; i < iteration ;i++){
        double x = hiprand_uniform_double(&state[index]);
        double y = hiprand_uniform_double(&state[index]);
        if(x*x + y*y <= 1.0)
            count++;
    }
    sum[index] = 4.0*count / iteration;
}

int main(void){
	// dim3 dimGrid(Nblock,1,1);  // Grid dimensions
	// dim3 dimBlock(Nthread,1,1);  // Block dimensions
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
    int i;
    int Athread = Nblock * Nthread;
    int iter = 1000000;
    double pi = 0.0;

    double *piSumHost =  (double *)malloc(Athread * sizeof(double));
    double *piSumDev;
    hipMalloc((void**)&piSumDev, Athread);
    hipMemset(piSumDev, 0, Athread);

    hiprandState *stateDev;
    hipMalloc((void **)&stateDev, Athread);
	hipEventRecord(start);
    setup <<<Ngrid, Nblock, Nthread>>> (stateDev);
    piEs  <<<Ngrid, Nblock, Nthread>>> (piSumDev, iter, stateDev);
	hipEventRecord(stop);
    hipMemcpy(piSumHost, piSumDev, Athread, hipMemcpyDeviceToHost);

	float runningTime = 0;
	hipEventElapsedTime(&runningTime, start, stop);
	printf("CUDA done, took %f ms\n", runningTime);

    for(i = 0; i<Athread; i++){
        pi += piSumHost[i];
    }
    pi /= Nblock;

    printf("Pi estimate = %.10lf\n", pi);

   free(piSumHost);
   hipFree(piSumDev);
   return 0;
}